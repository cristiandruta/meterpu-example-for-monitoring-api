/*
 * =====================================================================================
 *
 *       Filename:  lu_test.c
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016-04-25 11:41:27
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Lu Li, 
 *   Organization:  
 *
 * =====================================================================================
 */
#include <stdlib.h>

#include "MeterPU.h"


#ifdef SORT
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <vector>
#include <algorithm>
#include <iostream>

#endif

#define GPU_0 0

using namespace std;


#ifdef ENABLE_MF
#include <mf_api.h>
#endif

int main(int argc, const char *argv[])
{

#ifdef ENABLE_MF
    mf_api_new("http://mf.excess-project.eu:3030", "user_meterpu", "hello_app", NULL, "another_id");


    //define metric 
    mf_metric* metric = (mf_metric*)malloc(sizeof(mf_metric));
    
    metric->type = "meterpu";
    metric->name = "Power (milliW)";
#endif



#ifdef MEASURE
    using namespace meterpu;
    meter< NVML_Energy<GPU_0> > meter;
#endif 

#ifdef SORT
    std::vector<float> x(100000000);
    //std::vector<float> x(5);
    std::generate(x.begin(), x.end(), std::rand );
    thrust::device_vector<float> y(x.begin(), x.end() );
#endif

#ifdef MEASURE
    meter.start();
#endif 

#ifdef SORT
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
    thrust::transform(y.begin(), y.end(), y.begin(), thrust::negate<int>());
    hipDeviceSynchronize();
    thrust::sort(y.begin(), y.end() );
    hipDeviceSynchronize();
#else
    sleep(2);
#endif

#ifdef MEASURE
    meter.stop();
#endif

#ifdef SORT
    thrust::copy(y.begin(), y.end(), x.begin() );
    //std::for_each(x.begin(), x.end(), [](float const &i){ std::cout<<i<<" ";});
    //std::cout<<endl;

#endif



#ifdef MEASURE
    meter.calc();
#endif

#ifdef ENABLE_MF
    auto it=meter.measurement_controller_object.correctedPowerDB.begin();

    auto it_t=meter.measurement_controller_object.sampling_thread_controller.get_time_db().begin();
    auto end_t=meter.measurement_controller_object.sampling_thread_controller.get_time_db().end();

    char* response;

    for (; it_t!=end_t; ++it_t, ++it) {
    	    metric->timestamp = meterpu_mf_api_get_time_from_timespec(&(*it_t));
	    cout<<metric->timestamp<<endl;
	    metric->value = std::to_string(*it).c_str();
	    response = mf_api_update(metric);

    }

    cout<<response<<endl;
#endif

#ifdef MEASURE
    meter.show_meter_reading();
#endif



#ifdef ENABLE_MF
	
        free(metric);

	mf_api_clear();
#endif
	
	return 0;
}

